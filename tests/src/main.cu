/*
 * Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* Computation of eigenvalues of symmetric, tridiagonal matrix using
 * bisection.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>
#include <assert.h>

// includes, project
//#include <helper_functions.h>
//#include <hip/hip_runtime_api.h>

#include "viennacl/scalar.hpp"
#include "viennacl/vector.hpp"
#include "viennacl/matrix.hpp"
#include "viennacl/compressed_matrix.hpp"

#include "viennacl/linalg/eigenvalues/config.hpp"
#include "viennacl/linalg/eigenvalues/structs.hpp"
#include "viennacl/linalg/eigenvalues/matlab.hpp"
#include "viennacl/linalg/eigenvalues/util.hpp"
#include "viennacl/linalg/eigenvalues/gerschgorin.hpp"
#include "viennacl/linalg/eigenvalues/bisect_large.hpp"

#include "viennacl/linalg/eigenvalues/bisect_large.cuh"

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
bool runTest(int argc, char **argv);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char **argv)
{
    bool bQAResults = false;

    printf("Starting eigenvalues\n");

    bQAResults = runTest(argc, argv);
    printf("Test %s\n", bQAResults ? "Succeeded!" : "Failed!");

    exit(bQAResults ? EXIT_SUCCESS : EXIT_FAILURE);
}



////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
bool
runTest(int argc, char **argv)
{
    bool bCompareResult = false;
    // default
    unsigned int mat_size = 520;
    // flag if the matrix size is due to explicit user request
    unsigned int user_defined = 0;
    // desired precision of eigenvalues
    float  precision = 0.00001f;
    unsigned int iters_timing = 100;
    char  *result_file = "eigenvalues.dat";

    // set up input
    InputData input(argv[0], mat_size, user_defined);

    // compute Gerschgorin interval
    float lg = FLT_MAX;
    float ug = -FLT_MAX;
    computeGerschgorin(input.a, input.b + 1, mat_size, lg, ug);
    //computeGerschgorin(input.std_a, input.std_b_raw, mat_size, lg, ug);
    printf("Gerschgorin interval: %f / %f\n", lg, ug);


    // initialize memory for result
    ResultDataLarge result(mat_size);
    std::cout << "now initResultDataLargeMatrix" << std::endl;
    initResultDataLargeMatrix(result, mat_size);
/*
    // run the kernel
    computeEigenvaluesLargeMatrix(input, result, mat_size,
                                  precision, lg, ug,
                                  iters_timing);

    // get the result from the device and do some sanity checks
    // save the result if user specified matrix size
    bCompareResult = processResultDataLargeMatrix(input, result, mat_size, result_file,
                                                  user_defined, argv[0]);
*/
    // cleanup
    std::cout << "CleanupResultDataLargeMatrix!" << std::endl;
    cleanupResultDataLargeMatrix(result);

    std::cout << "cleanupInputData" << std::endl;
    input.cleanupInputData();

    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    
    std::cout << "hipDeviceReset" << std::endl;
    hipDeviceReset();

    return bCompareResult;
}

