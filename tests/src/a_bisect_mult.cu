#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* Computation of eigenvalues of symmetric, tridiagonal matrix using
 * bisection.
 */

#ifndef NDEBUG
  #define NDEBUG
#endif

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

// includes, project


#include "viennacl/scalar.hpp"
#include "viennacl/vector.hpp"
#include "viennacl/matrix.hpp"


#include "viennacl/linalg/eigenvalues/config.hpp"
#include "viennacl/linalg/eigenvalues/structs.hpp"
#include "viennacl/linalg/eigenvalues/matlab.hpp"
#include "viennacl/linalg/eigenvalues/util.hpp"
#include "viennacl/linalg/eigenvalues/gerschgorin.hpp"
#include "viennacl/linalg/eigenvalues/bisect_large.hpp"

#include "viennacl/linalg/eigenvalues/bisect_large.cuh"
#include "viennacl/linalg/eigenvalues/bisect_small.cuh"
#include "viennacl/linalg/eigenvalues/bisect_small.cu"

#include "viennacl/linalg/qr-method.hpp"

#define EPS 10.0e-4

//namespace viennacl
//{
  //namespace linalg
 // {
    ////////////////////////////////////////////////////////////////////////////////
    // declaration, forward
    bool runTest(int argc, char **argv);

    ////////////////////////////////////////////////////////////////////////////////
    // Program main
    ////////////////////////////////////////////////////////////////////////////////
    int
    main(int argc, char **argv)
    {
        bool bQAResults = false;

        printf("Starting eigenvalues\n");

        bQAResults = runTest(argc, argv);
        printf("Test %s\n", bQAResults ? "Succeeded!" : "Failed!");

        exit(bQAResults ? EXIT_SUCCESS : EXIT_FAILURE);
    }



    void
    initInputData(std::vector<float> &diagonal, std::vector<float> &superdiagonal, const unsigned int mat_size)
    {
     
      srand(278217421);
      bool randomValues = false;
      
      
      if(randomValues == true)
      {
        // Initialize diagonal and superdiagonal elements with random values
        for (unsigned int i = 0; i < mat_size; ++i)
        {
            diagonal[i] = (float)(2.0 * (((double)rand()
                                         / (double) RAND_MAX) - 0.5));
            superdiagonal[i] = (float)(2.0 * (((double)rand()
                                         / (double) RAND_MAX) - 0.5));
        }
      }
      
      else
      { 
        // Initialize diagonal and superdiagonal elements with modulo values
        // This will cause in many multiple eigenvalues.
        for(unsigned int i = 0; i < mat_size; ++i)
        {
           diagonal[i] = ((float)(i % 8)) - 4.5f;
           superdiagonal[i] = ((float)(i % 5)) - 4.5f;
        }
      }
      // the first element of s is used as padding on the device (thus the
      // whole vector is copied to the device but the kernels are launched
      // with (s+1) as start address
      superdiagonal[0] = 0.0f; 
    }
    
    template <typename NumericT>
    bool bisect(const std::vector<NumericT> & diagonal, const std::vector<NumericT> & superdiagonal, std::vector<NumericT> & eigenvalues, const unsigned int mat_size)
    {
        bool bCompareResult = false;
        // flag if the matrix size is due to explicit user request
        // desired precision of eigenvalues
        float  precision = 0.00001f;
        char  *result_file = "eigenvalues.dat";
        
        // set up input
        InputData input(diagonal, superdiagonal, mat_size);
        // compute Gerschgorin interval
        float lg = FLT_MAX;
        float ug = -FLT_MAX;
        //computeGerschgorin(input.a, input.b + 1, mat_size, lg, ug);
        computeGerschgorin(input.std_a, input.std_b, mat_size, lg, ug);
        printf("Gerschgorin interval: %f / %f\n", lg, ug);
        
        if (mat_size <= MAX_SMALL_MATRIX)
        {
          // initialize memory for result
          ResultDataSmall result(mat_size);

          // run the kernel
          computeEigenvaluesSmallMatrix(input, result, mat_size, lg, ug,
                                        precision);

          // get the result from the device and do some sanity checks,
          // save the result
          processResultSmallMatrix(input, result, mat_size, result_file);
          eigenvalues = result.std_eigenvalues;
          bCompareResult = true;
        }

        else
        {
          // initialize memory for result
          ResultDataLarge result(mat_size);
        
          // run the kernel
          computeEigenvaluesLargeMatrix(input, result, mat_size,
                                        lg, ug, precision);

         
           // get the result from the device and do some sanity checks
          // save the result if user specified matrix size
          bCompareResult = processResultDataLargeMatrix(input, result, mat_size, result_file);
                                                        
          eigenvalues = result.std_eigenvalues;                                  
          // cleanup
          std::cout << "CleanupResultData" << std::endl;
          result.cleanup();
        } //Large end
        return bCompareResult;
    }


    ////////////////////////////////////////////////////////////////////////////////
    //! Run a simple test for CUDA
    ////////////////////////////////////////////////////////////////////////////////
    bool
    runTest(int argc, char **argv)
    {
        bool bCompareResult = false;
        unsigned int mat_size = 950;
        
        std::vector<float> diagonal(mat_size);
        std::vector<float> superdiagonal(mat_size);
        std::vector<float> eigenvalues_bisect(mat_size);
        
        // Fill the diagonal and superdiagonal elements of the vector
        initInputData(diagonal, superdiagonal, mat_size);
        
        //Start the bisection algorithm
        std::cout << "Start the bisection algorithm" << std::endl;
        bCompareResult = bisect(diagonal, superdiagonal, eigenvalues_bisect, mat_size);
        
        // Exit if an error occured during the execution of the algorithm
        if (bCompareResult == false)
         return false;

        // The results of the bisection algorithm will be checked with the tql2 algorithm
        // Initialize Data for tql2 algorithm
        viennacl::matrix<float, viennacl::row_major> Q = viennacl::identity_matrix<float>(mat_size);
        std::vector<float> diagonal_tql(mat_size);
        std::vector<float> superdiagonal_tql(mat_size);
        diagonal_tql = diagonal;
        superdiagonal_tql = superdiagonal;
        
        // Start the tql2 algorithm
        std::cout << "Start the tql2 algorithm..." << std::endl; 
        viennacl::linalg::tql2(Q, diagonal_tql, superdiagonal_tql);  
        
        // Ensure that eigenvalues from tql2 algorithm are sorted in ascending order
        std::cout << "Start sorting..." << std::endl;
        std::sort(diagonal_tql.begin(), diagonal_tql.end());
        
        
        // Compare the results from the bisection algorithm with the results
        // from the tql2 algorithm.
        std::cout << "Start comparison..." << std::endl;
        for(uint i = 0; i < mat_size; i++)
        {
           if(std::abs(eigenvalues_bisect[i] - diagonal_tql[i]) > EPS)
           { 
	           std::cout << std::setprecision(8) << eigenvalues_bisect[i] << "  != " << diagonal_tql[i] << "\n";
	           return false;
           }  	
        }
        
        std::cout << "mat_size = " << mat_size << std::endl;
        // Print the results.
        for (unsigned int i = 0; i < mat_size; ++i)
        {
          std::cout << "Eigenvalue " << i << ":  \tbisect: " << std::setprecision(8) << eigenvalues_bisect[i] << "\ttql2: " << diagonal_tql[i] << std::endl;
        }
        
        
        // hipDeviceReset causes the driver to clean up all state. While
        // not mandatory in normal operation, it is good practice.  It is also
        // needed to ensure correct operation when the application is being
        // profiled. Calling hipDeviceReset causes all profile data to be
        // flushed before the application exits
        
        std::cout << "hipDeviceReset" << std::endl;
        hipDeviceReset();

        return bCompareResult;
    }
//  }
//}
