/*
 * Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* Computation of eigenvalues of a small symmetric, tridiagonal matrix */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>
#include  <algorithm>

// includes, project

#include "config.hpp"
#include "structs.hpp"
#include "matlab.hpp"

// includes, kernels
#include "bisect_kernel_small.cuh"

// includes, file
#include "bisect_small.cuh"

////////////////////////////////////////////////////////////////////////////////
//! Determine eigenvalues for matrices smaller than MAX_SMALL_MATRIX
//! @param TimingIterations  number of iterations for timing
//! @param  input  handles to input data of kernel
//! @param  result handles to result of kernel
//! @param  mat_size  matrix size
//! @param  lg  lower limit of Gerschgorin interval
//! @param  ug  upper limit of Gerschgorin interval
//! @param  precision  desired precision of eigenvalues
//! @param  iterations  number of iterations for timing
////////////////////////////////////////////////////////////////////////////////
void
computeEigenvaluesSmallMatrix(const InputData &input, ResultDataSmall &result,
                              const unsigned int mat_size,
                              const float lg, const float ug,
                              const float precision)
{
    

    dim3  blocks(1, 1, 1);
    dim3  threads(MAX_THREADS_BLOCK_SMALL_MATRIX, 1, 1);

    bisectKernel<<< blocks, threads >>>(input.g_a, input.g_b, mat_size,
                                        result.g_left, result.g_right,
                                        result.g_left_count,
                                        result.g_right_count,
                                        lg, ug, 0, mat_size,
                                        precision
                                       );
    

    checkCudaErrors(hipDeviceSynchronize());

//    getLastCudaError("Kernel launch failed");

}


////////////////////////////////////////////////////////////////////////////////
//! Process the result obtained on the device, that is transfer to host and
//! perform basic sanity checking
//! @param  input  handles to input data
//! @param  result  handles to result data
//! @param  mat_size   matrix size
//! @param  filename  output filename
////////////////////////////////////////////////////////////////////////////////
void
processResultSmallMatrix(const InputData &input, ResultDataSmall &result,
                         const unsigned int mat_size,
                         const char *filename)
{

    const unsigned int mat_size_f = sizeof(float) * mat_size;
    const unsigned int mat_size_ui = sizeof(unsigned int) * mat_size;

    // copy data back to host
    float *left = (float *) malloc(mat_size_f);
    unsigned int *left_count = (unsigned int *) malloc(mat_size_ui);

    checkCudaErrors(hipMemcpy(left, result.g_left, mat_size_f,
                               hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(left_count, result.g_left_count, mat_size_ui,
                               hipMemcpyDeviceToHost));

    float *eigenvalues = (float *) malloc(mat_size_f);

    for (unsigned int i = 0; i < mat_size; ++i)
    {
        result.std_eigenvalues[left_count[i]] = left[i];
    }
    
    /* for (unsigned int i = 0; i < mat_size; ++i)
    {
        std::cout << "Eigenvalues small: " << result.std_eigenvalues[i] << std::endl;
    }*/
    
    
   

    // save result in matlab format
   // writeTridiagSymMatlab(filename, input.a, input.b+1, eigenvalues, mat_size);

    freePtr(left);
    freePtr(left_count);
    freePtr(eigenvalues);
}
